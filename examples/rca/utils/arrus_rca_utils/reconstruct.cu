#include "hip/hip_runtime.h"
#include <cupy/complex.cuh>
#include <hip/hip_math_constants.h>

#define HIP_PI_F 3.141592654f

extern "C"
__global__ void
iqRaw2Hri(complex<float> *iqLri, const complex<float> *iqRaw,
          const int nTx, const int nSamp, const int nRx,
          const float *zPixGrid, const int nZPix,
          const float *xPixGrid, const int nXPix,
          const float *yPixGrid, const int nYPix,
          float const sos, float const fs, float const fn,
          const float *txFoc, const float *txAng,
          const float *txApCent,
          const float txProbePitch, const int txProbeNElem, const unsigned char txProbeOri,
          const float rxProbePitch, const int rxProbeNElem, const unsigned char rxProbeOri,
          const float *txApFstElemPos, const float *txApLstElemPos,
          const int *rxApOrigElem,
          const float minRxTang, const float maxRxTang,
          float const initDel,
          const float nSigma) {

    int z = blockIdx.x * blockDim.x + threadIdx.x;
    int x = blockIdx.y * blockDim.y + threadIdx.y;
    int y = blockIdx.z * blockDim.z + threadIdx.z;

    if(z >= nZPix || x >= nXPix || y >= nYPix) {
        return;
    }
    float zPix = zPixGrid[z];
    float xPix = xPixGrid[x];
    float yPix = yPixGrid[y];

    float txLateralPix, rxLateralPix;
    float rxElem;

    txLateralPix = txProbeOri == 0 ? xPix : yPix;

    int iElem, offset;
    float interpWgh;
    float txDist, rxDist, rxTang, txApod, rxApod, time, iSamp;
    float modSin, modCos, pixWgh;
    const float omega = 2 * HIP_PI_F * fn;
    const float sosInv = 1 / sos;
    const float twoSigSqrInv = nSigma * nSigma * 0.5f;
    const float rngRxTangInv = 2 / (maxRxTang - minRxTang); // inverted half range
    const float centRxTang = (maxRxTang + minRxTang) * 0.5f;
    complex<float> pix(0.0f, 0.0f), samp(0.0f, 0.0f), modFactor;

    iqLri[z + x*nZPix + y*nZPix*nXPix] = complex<float>(0.0f, 0.0f);

    for(int iTx = 0; iTx < nTx; ++iTx) {
        int txOffset = iTx*nSamp*nRx;
        if(!isinf(txFoc[iTx])) {
            /* STA */

            float zFoc = txFoc[iTx]*cosf(txAng[iTx]);
            float lateralFoc = txApCent[iTx] + txFoc[iTx] * sinf(txAng[iTx]);

            float pixFocArrang;
            if(txFoc[iTx] <= 0.0f) {
                /* Virtual Point Source BEHIND probe surface */
                // Valid pixels are assumed to be always in front of the focal point (VSP)
                pixFocArrang = 1.0f;
            } else {
                /* Virtual Point Source IN FRONT OF probe surface */
                // Projection of the Foc-Pix vector on the ApCent-Foc vector (dot product) ...
                // to determine if the pixel is behind (-) or in front of (+) the focal point (VSP).
                pixFocArrang = (((zPix - zFoc) * zFoc +
                                 (txLateralPix - lateralFoc) * (lateralFoc-txApCent[iTx])) >= 0.f) ? 1.f : -1.f;
            }
            txDist = hypotf(zPix-zFoc, txLateralPix-lateralFoc);
            txDist *= pixFocArrang; // Compensation for the Pix-Foc arrangement
            txDist += txFoc[iTx]; // Compensation for the reference time being the moment when txApCent fires.

            // Projections of Foc-Pix vector on the rotated Foc-ApEdge vectors (dot products) ...
            // to determine if the pixel is in the sonified area (dot product >= 0).
            // Foc-ApEdgeFst vector is rotated left, Foc-ApEdgeLst vector is rotated right.
            txApod = (((-(txApFstElemPos[iTx] - lateralFoc) * (zPix-zFoc) +
                        (-zFoc)*(txLateralPix - lateralFoc)) * pixFocArrang >= 0.f) &&
                      (((txApLstElemPos[iTx] - lateralFoc) * (zPix-zFoc) -
                        (-zFoc)*(txLateralPix - lateralFoc)) * pixFocArrang >= 0.f)) ? 1.f : 0.f;
        } else {
            /* PWI */
            txDist = zPix * cosf(txAng[iTx]) + (txLateralPix - txApCent[iTx]) * sinf(txAng[iTx]);
            // Projections of ApEdge-Pix vector on the rotated unit vector of tx direction (dot products) ...
            // to determine if the pixel is in the sonified area (dot product >= 0).
            // For ApEdgeFst, the vector is rotated left, for ApEdgeLst the vector is rotated right.
            txApod = (((-zPix * sinf(txAng[iTx]) +
                        (txLateralPix - txApFstElemPos[iTx]) * cosf(txAng[iTx])) >= 0.f) &&
                      ((zPix * sinf(txAng[iTx]) -
                        (txLateralPix - txApLstElemPos[iTx]) * cosf(txAng[iTx])) >= 0.f)) ? 1.f : 0.f;
        }
        pixWgh = 0.0f;
        pix.real(0.0f);
        pix.imag(0.0f);

        if(txApod != 0.0f) {
            rxElem = -((float)(rxProbeNElem-1) / 2.0f)*rxProbePitch;
            for(int iRx = 0; iRx < nRx; ++iRx) {
                iElem = iRx + rxApOrigElem[iTx];
                rxElem += rxProbePitch;
                if(iElem < 0 || iElem >= rxProbeNElem) continue;
                rxLateralPix = rxProbeOri == 0 ? xPix : yPix;
                rxDist = hypotf(rxLateralPix - rxElem, zPix);
                rxTang = __fdividef(rxLateralPix - rxElem, zPix);
                if(rxTang < minRxTang || rxTang > maxRxTang) continue;
                rxApod = (rxTang-centRxTang) * rngRxTangInv;
                rxApod = __expf(-rxApod*rxApod*twoSigSqrInv);
                time = (txDist+rxDist)*sosInv + initDel;
                iSamp = time*fs;
                if(iSamp < 0.0f || iSamp >= static_cast<float>(nSamp - 1)) {
                    continue;
                }
                offset = txOffset + iRx*nSamp;
                interpWgh = modff(iSamp, &iSamp);
                int intSamp = int(iSamp);
                __sincosf(omega*time, &modSin, &modCos);
                complex<float> modFactor = complex<float>(modCos, modSin);
                samp = iqRaw[offset+intSamp]*(1-interpWgh) + iqRaw[offset+intSamp+1]*interpWgh;
                pix += samp*modFactor*rxApod;
                pixWgh += rxApod;
            }
        }
        if(pixWgh != 0.0f) {
            iqLri[z + x*nZPix + y*nZPix*nXPix] += pix/pixWgh*txApod;
        }
    }
}
